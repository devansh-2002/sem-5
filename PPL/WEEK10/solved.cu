//Matrix multiplication of 4x4 matrix
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#define BLOCK_w 2
#define TILE_w 2
#define w 4

__global__ void MatMulElementThreadShared(int *a, int *b, int *c) {
__shared__ int MDs[TILE_w][TILE_w];
__shared__ int NDs[TILE_w][TILE_w];
int m;
int bx=blockIdx.x; int by=blockIdx.y;
int tx=threadIdx.x; int ty=threadIdx.y;
int Row=by*TILE_w + ty;
int Col= bx*TILE_w + tx;
int Pvalue=0;
for(m=0; m<w/TILE_w; m++){
MDs[ty][tx]=a[Row*w+m*TILE_w+tx];
NDs[ty][tx]=b[(m*TILE_w+ty)*w+Col];
__syncthreads();
for (int k = 0; k < TILE_w; k++){
	Pvalue += MDs[ty][k]*NDs[k][tx];
				}
	__syncthreads();
	}
	c[Row*w+Col] = Pvalue;
}

int main() {
int *matA, *matB, *matProd;
int *da, *db, *dc;
printf("\n== Enter elements of Matrix A (4x4) ==\n");
matA = (int*)malloc(sizeof(int) * w * w);
for(int i = 0; i < w * w; i++)
{
scanf("%d", &matA[i]);
}
printf("\n== Enter elements of Matrix B (4x4) ==\n");
matB = (int*)malloc(sizeof(int) * w * w);
for(int i = 0; i < w * w; i++)
{
scanf("%d", &matB[i]);
}
matProd = (int*)malloc(sizeof(int) * w * w);
hipMalloc((void **) &da, sizeof(int) * w * w);
hipMalloc((void **) &db, sizeof(int) * w * w);
hipMalloc((void **) &dc, sizeof(int) * w * w);
hipMemcpy(da, matA, sizeof(int) * w *w, hipMemcpyHostToDevice);
hipMemcpy(db, matB, sizeof(int) * w *w, hipMemcpyHostToDevice);
int NumBlocks = w / BLOCK_w;
dim3 grid_conf (NumBlocks, NumBlocks);
dim3 block_conf (BLOCK_w, BLOCK_w);
MatMulElementThreadShared<<<grid_conf, block_conf>>>(da, db, dc);
hipMemcpy(matProd,dc,sizeof(int)* w *w,hipMemcpyDeviceToHost);
printf("\n-=Result of Addition=-\n");
printf("\n");
for (int i = 0; i < w; i++ ) {
for (int j = 0; j < w; j++) {
printf("%6d ", matProd[i * w + j]);
}printf("\n");
}
hipFree(da);
hipFree(db);
hipFree(dc);
free(matA);
free(matB);
free(matProd);
return 0;
}
