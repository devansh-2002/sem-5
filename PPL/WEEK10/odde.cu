
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void oddEvenSort(int *arr, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int phase, i, temp;

    for (phase = 0; phase < n; phase++)
    {
        if (phase % 2 == 0)
        { // Even phase
            if (tid % 2 == 0 && tid < n - 1)
            {
                if (arr[tid] > arr[tid + 1])
                {
                    // Swap adjacent elements
                    temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        else
        { // Odd phase
            if (tid % 2 == 1 && tid < n - 1)
            {
                if (arr[tid] > arr[tid + 1])
                {
                    // Swap adjacent elements
                    temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        __syncthreads(); // Synchronize threads before next phase
    }
}

int main()
{
    const int n = 8;
    int hostArray[n] = {5, 2, 9, 3, 1, 6, 8, 4};
    int *deviceArray;

    // Allocate memory on the GPU
    hipMalloc((void **)&deviceArray, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(deviceArray, hostArray, n * sizeof(int), hipMemcpyHostToDevice);

    // Define the block and grid dimensions
    int block_size = 4;
    int grid_size = (n + block_size - 1) / block_size;

    // Launch the kernel
    oddEvenSort<<<grid_size, block_size>>>(deviceArray, n);

    // Copy data back from device to host
    hipMemcpy(hostArray, deviceArray, n * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(deviceArray);

    // Print sorted array
    printf("Sorted array: ");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", hostArray[i]);
    }
    printf("\n");

    return 0;
}
