#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MASK_SIZE 3

__constant__ int mask[MASK_SIZE];

__global__ void convolution(int *input, int *output, int width)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < width)
    {
        int res = 0;
        for (int i = 0; i < MASK_SIZE; i++)
        {
            int idx = tid - MASK_SIZE / 2 + i;
            if (idx >= 0 && idx < width)
            {
                res += input[idx] * mask[i];
            }
        }
        output[tid] = res;
    }
}

int main()
{
    int width, *input, *output, *d_input, *d_output;
    printf("Enter the width:\n");
    scanf("%d", &width);
    input = (int *)malloc(sizeof(int) * width);
    output = (int *)malloc(sizeof(int) * width);

    printf("Enter the array elements:\n");
    for (int i = 0; i < width; i++)
        scanf("%d", &input[i]);

    printf("Enter the mask elements:\n");
    int maskElements[MASK_SIZE];
    for (int i = 0; i < MASK_SIZE; i++)
        scanf("%d", &maskElements[i]);

    // Copy the mask to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(mask), maskElements, MASK_SIZE * sizeof(int));

    hipMalloc((void **)&d_input, sizeof(int) * width);
    hipMalloc((void **)&d_output, sizeof(int) * width);
    hipMemcpy(d_input, input, sizeof(int) * width, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (width + blockSize - 1) / blockSize;
    convolution<<<gridSize, blockSize>>>(d_input, d_output, width);
    hipMemcpy(output, d_output, sizeof(int) * width, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < width; i++)
    {
        printf("%d ", output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);
    free(input);
    free(output);

    return 0;
}
