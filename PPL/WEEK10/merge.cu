#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int N = 8; // The number of elements in the array

__global__ void merge(int *arr, int *temp, int left, int middle, int right)
{
    int i = left;
    int j = middle + 1;

    for (int k = left; k <= right; k++)
    {
        if (i <= middle && (j > right || arr[i] <= arr[j]))
        {
            temp[k] = arr[i];
            i++;
        }
        else
        {
            temp[k] = arr[j];
            j++;
        }
    }

    for (int k = left; k <= right; k++)
    {
        arr[k] = temp[k];
    }
}

__global__ void mergeSort(int *arr, int *temp, int n)
{
    for (int currentSize = 1; currentSize < n; currentSize *= 2)
    {
        for (int leftStart = 0; leftStart < n - 1; leftStart += 2 * currentSize)
        {
            int middle = min(leftStart + currentSize - 1, n - 1);
            int rightEnd = min(leftStart + 2 * currentSize - 1, n - 1);
        }
    }
}
__host__ void hostMergeSort(int *deviceArray, int *deviceTemp, int n)
{
    for (int currentSize = 1; currentSize < n; currentSize *= 2)
    {
        for (int leftStart = 0; leftStart < n - 1; leftStart += 2 * currentSize)
        {
            int middle = min(leftStart + currentSize - 1, n - 1);
            int rightEnd = min(leftStart + 2 * currentSize - 1, n - 1);
            merge<<<1, 1>>>(deviceArray, deviceTemp, leftStart, middle, rightEnd);
            hipDeviceSynchronize(); // Wait for the kernel to finish
        }
    }
}

int main()
{
    int hostArray[N] = {5, 2, 9, 3, 1, 6, 8, 4};
    int *deviceArray, *deviceTemp;

    // Allocate memory on the GPU
    hipMalloc((void **)&deviceArray, N * sizeof(int));
    hipMalloc((void **)&deviceTemp, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(deviceArray, hostArray, N * sizeof(int), hipMemcpyHostToDevice);

    // Call the hostMergeSort function
    hostMergeSort(deviceArray, deviceTemp, N);

    // Copy data back from device to host
    hipMemcpy(hostArray, deviceArray, N * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(deviceArray);
    hipFree(deviceTemp);

    // Print sorted array
    printf("Sorted array: ");
    for (int i = 0; i < N; i++)
    {
        printf("%d ", hostArray[i]);
    }
    printf("\n");

    return 0;
}
