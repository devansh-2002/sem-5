#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


__global__ void calculateSine(float *angles, float *sineResults, int numAngles) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numAngles) {
        sineResults[tid] = sinf(angles[tid]);
    }
}
int main() {
    int numAngles = 5; 
    float h_angles[] = {41.0, 42.0, 43.0, 44.0, 45.0}; 
    float h_sineResults[numAngles]; 

    float *d_angles, *d_sineResults;
    int size = numAngles * sizeof(float); 

    hipMalloc((void **)&d_angles, size);
    hipMalloc((void **)&d_sineResults, size);

    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);
    int blockSize = 256;
    int numBlocks = (numAngles + blockSize - 1) / blockSize;

    calculateSine<<<numBlocks, blockSize>>>(d_angles, d_sineResults, numAngles);

    hipMemcpy(h_sineResults, d_sineResults, size, hipMemcpyDeviceToHost);

    printf("Input Angles (in radians):\n");
    for (int i = 0; i < numAngles; i++) {
        printf("%.2f ", h_angles[i]);
    }
    printf("\n\nSine Results:\n");
    for (int i = 0; i < numAngles; i++) {
        printf("%.4f ", h_sineResults[i]);
    }
    printf("\n");

    hipFree(d_angles);
    hipFree(d_sineResults);
    return 0;
}






