// convolution operation on one dimensional input array N of size width using a mask array M of size mask_width to produce the resultant one dimensional array P of size width.

#include<stdio.h>	
#include<math.h>
#include "hip/hip_runtime.h"

__global__ void convulution(int *n, int *m, int *p, int len, int maskLength)
{    
    int tid=(blockIdx.x*blockDim.x)+threadIdx.x;
    int j;
    float pvalue=0;
    int start=tid-(maskLength/2);
    for(j=0;j<maskLength;j++)
    {
        if(start+j>=0 && start+j<=len)
        { 
            pvalue+=n[start+j]*m[j];
        }
    }
    p[tid]=pvalue;
}		
int main(void) 
{
    int len,mask_width;
    printf("Enter size of array:"); 
    scanf("%d",&len);
    printf("Enter mask width:");
    scanf("%d",&mask_width);
    int size = len * sizeof(int);
    int a[len],m[mask_width], p[len], i;
    int *d_a, *d_m, *d_p;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_m, size);
    hipMalloc((void **)&d_p, size);
    printf("\nEnter array:\n");
    for(i=0;i<len;i++)
    {

        scanf("%d",&a[i]);
    }
    printf("Enter mask\n");
    for(i=0;i<mask_width;i++)
    {

        scanf("%d",&m[i]);
    }
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_m, &m, size, hipMemcpyHostToDevice);
    dim3 dimGrid(ceil(len/256.0),1,1);
    dim3 dimBlock(256,1,1);
    convulution<<<dimGrid,dimBlock>>>(d_a, d_m, d_p,len,mask_width);	
    hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(i=0;i<len;i++)
    printf("%d ",p[i]);
    hipFree(d_a);
    hipFree(d_m);
    hipFree(d_p);
    return 0;
}
