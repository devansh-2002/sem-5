#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_w 2
#define w 4

__device__ int getTid(){
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int rowInBlock = threadIdx.y;
    int colInBlock = threadIdx.x;
    int globalRow = blockRow * blockDim.y + rowInBlock;
    int globalCol = blockCol * blockDim.x + colInBlock;
    return (globalRow*w + globalCol);
}

__global__ void matMul(int* a,int* b, int* c){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int sum = 0;
    for (int k = 0; k < w; k++)
        sum += a[row*w + k] * b[k*w + col];
    c[row*w + col] = sum;
} 

int main(){
    int* matA,*matB,*matC,*da,*db,*dc;
    printf("Enter the elements of (4x4) matrix A:\n");
    matA = (int*)malloc(sizeof(int)*w*w);
    for (int i = 0; i < w*w; i++)
        scanf("%d",&matA[i]);
    printf("Enter the elements of (4x4) matrix B:\n");
    matB = (int*)malloc(sizeof(int)*w*w);
    for (int i = 0; i < w*w; i++)
        scanf("%d",&matB[i]);
    matC = (int*)malloc(sizeof(int)*w*w);
    hipMalloc((void**) &da,sizeof(int)*w*w);
    hipMalloc((void**) &db,sizeof(int)*w*w);
    hipMalloc((void**) &dc,sizeof(int)*w*w);
    hipMemcpy(da,matA,sizeof(int)*w*w,hipMemcpyHostToDevice);
    hipMemcpy(db,matB,sizeof(int)*w*w,hipMemcpyHostToDevice);
    dim3 grid_conf(w/TILE_w,w/TILE_w);
    dim3 block_conf(TILE_w,TILE_w);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(matC,dc,sizeof(int)*w*w,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < w; i++){
        for (int j = 0; j < w; j++){
            printf("%6d ",matC[i*w + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}
