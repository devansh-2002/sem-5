#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_w 2
#define w 4
#define MASK_w 3

__global__ void convolution(int *input, int *mask, int *output) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    for (int i = 0; i < MASK_w; i++) {
        for (int j = 0; j < MASK_w; j++) {
            int inputRow = row + i - MASK_w / 2;
            int inputCol = col + j - MASK_w / 2;

            if (inputRow >= 0 && inputRow < w && inputCol >= 0 && inputCol < w) {
                sum += input[inputRow * w + inputCol] * mask[i * MASK_w + j];
            }
        }
    }

    output[row * w + col] = sum;
}

int main() {
    int *input, *mask, *output, *d_input, *d_mask, *d_output;

    printf("Enter the elements of (4x4) input matrix:\n");
    input = (int*)malloc(sizeof(int) * w * w);
    for (int i = 0; i < w * w; i++) {
        scanf("%d", &input[i]);
    }

    printf("Enter the elements of (3x3) mask matrix:\n");
    mask = (int*)malloc(sizeof(int) * MASK_w * MASK_w);
    for (int i = 0; i < MASK_w * MASK_w; i++) {
        scanf("%d", &mask[i]);
    }

    output = (int*)malloc(sizeof(int) * w * w);
    hipMalloc((void**)&d_input, sizeof(int) * w * w);
    hipMalloc((void**)&d_mask, sizeof(int) * MASK_w * MASK_w);
    hipMalloc((void**)&d_output, sizeof(int) * w * w);

    hipMemcpy(d_input, input, sizeof(int) * w * w, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, sizeof(int) * MASK_w * MASK_w, hipMemcpyHostToDevice);

    dim3 grid_conf(w / TILE_w, w / TILE_w);
    dim3 block_conf(TILE_w, TILE_w);

    convolution<<<grid_conf, block_conf>>>(d_input, d_mask, d_output);

    hipMemcpy(output, d_output, sizeof(int) * w * w, hipMemcpyDeviceToHost);

    printf("Result of Convolution:\n");
    for (int i = 0; i < w; i++) {
        for (int j = 0; j < w; j++) {
            printf("%6d ", output[i * w + j]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
    free(input);
    free(mask);
    free(output);

    return 0;
}
