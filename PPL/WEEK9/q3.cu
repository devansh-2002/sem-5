#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include<hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "stb_image/stb_image.h"
#include "stb_image/stb_image_write.h"

int main() {
    // Load the input image
    int width, height, channels;
    unsigned char* input_image = stbi_load("Lena.jpeg", &width, &height, &channels, 0);

    if (input_image == NULL) {
        printf("Failed to load the input image.\n");
        return 1;
    }

    // Create an output image of the same dimensions
    unsigned char* output_image = (unsigned char*)malloc(width * height * channels);

    // Embossing kernel
    int kernel[3][3] = {
        {-2, -1, 0},
        {-1,  1, 1},
        { 0,  1, 2}
    };

    // Apply embossing
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            for (int c = 0; c < channels; c++) {
                int embossed_pixel = 0;
                for (int i = -1; i <= 1; i++) {
                    for (int j = -1; j <= 1; j++) {
                        int pixel_value = input_image[((y + i) * width + (x + j)) * channels + c];
                        embossed_pixel += pixel_value * kernel[i + 1][j + 1];
                    }
                }
                embossed_pixel = embossed_pixel > 255 ? 255 : (embossed_pixel < 0 ? 0 : embossed_pixel);
                output_image[(y * width + x) * channels + c] = embossed_pixel;
            }
        }
    }

    // Save the embossed image
    stbi_write_jpg("output_embossed.jpg", width, height, channels, output_image, 100);

    // Free memory
    stbi_image_free(input_image);
    free(output_image);

    return 0;
}
