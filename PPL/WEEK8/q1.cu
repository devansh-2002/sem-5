#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void csr(int num_rows, int* data, int* col_index, int* row_ptr, int* x, int* y) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < num_rows) {
        int res = 0;
        int start = row_ptr[row];
        int stop = row_ptr[row + 1];
        for (int i = start; i < stop; i++) {
            res += data[i] * x[col_index[i]];
        }
        y[row] = res;
    }
}

int main() {
    int m, n;
    printf("Enter the dimensions of the matrix:\n");
    scanf("%d %d", &m, &n);
    printf("Enter the sparse matrix:\n");
    int *mat = (int*)malloc(sizeof(int) * m * n);
    int non_zero_count = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            scanf("%d", &mat[i * n + j]);
            if (mat[i * n + j] != 0) {
                non_zero_count++;
            }
        }
    }

    int* data = (int*)malloc(sizeof(int) * non_zero_count);
    int* col_index = (int*)malloc(sizeof(int) * non_zero_count);
    int* x = (int*)malloc(sizeof(int) * n);
    printf("Enter the elements of the vector x:\n");
    for (int i = 0; i < n; i++) {
        scanf("%d", &x[i]);
    }
    int* row_ptr = (int*)malloc(sizeof(int) * (m + 1));
    int* y = (int*)malloc(sizeof(int) * m);
    int id = 0;
    row_ptr[0] = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int k = i * n + j;
            if (mat[k] != 0) {
                data[id] = mat[k];
                col_index[id] = j;
                id += 1;
            }
        }
        row_ptr[i + 1] = id;
    }
    int* d_data, *d_col_index, *d_row_ptr, *d_x, *d_y;
    hipMalloc((void**)&d_data, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_col_index, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (m + 1) * sizeof(int));
    hipMalloc((void**)&d_x, n * sizeof(int));
    hipMalloc((void**)&d_y, m * sizeof(int));
    hipMemcpy(d_data, data, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);
    csr<<<1, m>>>(m, d_data, d_col_index, d_row_ptr, d_x, d_y);
    hipMemcpy(y, d_y, m * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < m; i++) {
        printf("%d ", y[i]);
    }
    printf("\n");
    return 0;
}
